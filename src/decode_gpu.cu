#include "hip/hip_runtime.h"
#include "decode_gpu.cuh"
#include "error.h"
#include "error.cuh"
#include "misc.h"

#include <math.h>
#include <vector>
#include <float.h>

// https://stackoverflow.com/questions/17399119/how-do-i-use-atomicmax-on-floating-point-values-in-cuda
__device__ __forceinline__ static float atomicMaxFloat (float * addr, float value) {
    float old;
    old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
         __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));

    return old;
}

__global__ void bwd_scan(
	const DTYPE_GPU *scores_in,
	DTYPE_GPU *out,
	const int T,
	const int N,
	const int num_states,
	const int num_states_per_thread
) {
	uint64_t chunk = blockIdx.x + (blockIdx.y * gridDim.x);
	uint64_t thread_idx = threadIdx.x + (threadIdx.y * blockDim.x);
	uint64_t state_begin = thread_idx * num_states_per_thread;
	uint64_t state_end = state_begin + num_states_per_thread;

	if (chunk >= N || state_begin >= num_states) {
		return;
	}

	const uint64_t k_num_bases = 4;
    const uint64_t k_num_transitions = k_num_bases + 1;
    const DTYPE_GPU k_fixed_stay_score = 2.0f;

    const uint64_t ts_states = num_states * k_num_bases;

    const DTYPE_GPU* const chunk_in = scores_in + chunk * ts_states; // should be half DTYPE_GPU (for GPU impl)
    DTYPE_GPU* const chunk_out = out + chunk * (T+1) * num_states;
    DTYPE_GPU* const alpha_init = chunk_out + num_states * T;
    for (uint64_t state = state_begin; state < state_end; ++state) {
        alpha_init[state] = 0.0f;
    }

    for (uint64_t ts = 0; ts < T; ++ts) {
        __syncthreads();
        const DTYPE_GPU* const ts_in = chunk_in + N * ts_states * (T - ts - 1);
        DTYPE_GPU* const ts_alpha_in = alpha_init - num_states * ts;
        DTYPE_GPU* const ts_alpha_out = ts_alpha_in - num_states;

        for (uint64_t state = state_begin; state < state_end; ++state) {
            const uint64_t stay_state_idx = state;
            const uint64_t step_state_idx_a = (state * k_num_bases) % num_states;
            const uint64_t step_trans_idx_a = step_state_idx_a * k_num_bases +
                ((state * k_num_bases) / num_states);

            DTYPE_GPU vals[k_num_transitions];
            vals[0] = ts_alpha_in[stay_state_idx] + k_fixed_stay_score;
            DTYPE_GPU max_val = vals[0];
            for (uint64_t base = 0; base < k_num_bases; ++base) {
                vals[base + 1] = ts_alpha_in[step_state_idx_a + base] +
                    ts_in[step_trans_idx_a + base * k_num_bases];
                max_val = max_val > vals[base + 1] ? max_val : vals[base + 1];
            }
            DTYPE_GPU sum = 0.0f;
            for (uint64_t i = 0; i < k_num_transitions; ++i) {
                sum += __expf(vals[i] - max_val);
            }
            ts_alpha_out[state] = max_val + __logf(sum);
        }
    }
}

__global__ void fwd_post_scan(
    const DTYPE_GPU *scores_in,
    const DTYPE_GPU *bwd,
    DTYPE_GPU *out,
    const uint64_t _T,
    const uint64_t N,
    const uint64_t num_states,
    const int num_states_per_thread
) {
    uint64_t chunk = blockIdx.x + (blockIdx.y * gridDim.x);
	uint64_t thread_idx = threadIdx.x + (threadIdx.y * blockDim.x);
	uint64_t state_begin = thread_idx * num_states_per_thread;
	uint64_t state_end = state_begin + num_states_per_thread;

	if (chunk >= N || state_begin >= num_states) {
		return;
	}

    const uint64_t T = _T+1; 
    constexpr uint64_t k_num_bases = 4;
    constexpr uint64_t k_num_transitions = k_num_bases + 1;
    constexpr DTYPE_GPU k_fixed_stay_score = 2.0f;
    
    const uint64_t kMsb = num_states / k_num_bases;
    const uint64_t ts_states = num_states * k_num_bases;

    constexpr uint64_t k_max_states = 1024;
    __shared__ DTYPE_GPU fwd_vals[k_max_states];
    __shared__ DTYPE_GPU exp_vals[k_max_states];
    __shared__ DTYPE_GPU exp_sum;
    __shared__ DTYPE_GPU max_val;
    max_val = FLT_MIN;

    // This batch element's scores.
    const DTYPE_GPU* const chunk_scores = scores_in + chunk * ts_states;

    // Alternating forward guide buffers used for successive time steps.
    __shared__ DTYPE_GPU ts_fwd[2][k_max_states]; // threadgroup

    // The forward guide input for the first step is 0.
    for (uint64_t state = state_begin; state < state_end; ++state) {
        ts_fwd[0][state] = 0.0f;
    }
    __syncthreads();

    for (uint64_t ts = 0; ts < T; ++ts) {
        // We read forward guide values written to TG memory in the previous step as
        // inputs to this step.  However, there has already been a TG barrier since
        // they were written.
        const uint64_t ts_idx = (chunk * T + ts) * num_states;

        // This time step's scores.
        const DTYPE_GPU* const ts_scores = chunk_scores + N * ts_states * ts;

        // Alternating TG buffer twiddling.
        const DTYPE_GPU* const ts_alpha_in = ts_fwd[ts & 1];
        DTYPE_GPU* const ts_alpha_out = ts_fwd[(ts & 1) ^ 1];

        // Calculate the next time step's forward guide from this time step's scores
        // and forward guide.  It's written to threadgroup memory for use in the
        // next iteration.
        for (uint64_t state = state_begin; state < state_end; ++state) {
            const uint64_t stay_state_idx = state;
            const uint64_t step_state_idx_a = state / k_num_bases;
            const uint64_t step_trans_idx_a = state * k_num_bases;
            DTYPE_GPU vals[k_num_transitions];
            DTYPE_GPU fwd_max_val = vals[0] = ts_alpha_in[stay_state_idx] + k_fixed_stay_score;
            for (uint64_t base = 0; base < k_num_bases; ++base) {
                // todo: this is a bandaid for indexing past the actual T dimension of scores
                // need to verify with actual MetalTxCaller impl output,
                // otherwise output remains exactly the same for this impl whether it indexes past or not
                DTYPE_GPU ts_score = ts < _T ? ts_scores[step_trans_idx_a + base] : 0.0f;

                vals[base + 1] = ts_alpha_in[step_state_idx_a + base * kMsb] + ts_score;
                fwd_max_val = fwd_max_val > vals[base + 1] ? fwd_max_val : vals[base + 1];
            }
            DTYPE_GPU fwd_sum = 0.0f;
            for (uint64_t i = 0; i < k_num_transitions; ++i) {
                fwd_sum += exp(vals[i] - fwd_max_val);
            }
            ts_alpha_out[state] = fwd_max_val + __logf(fwd_sum);

            // Load the forward guide value calculated in the last time step for use
            // in this time step's posterior probability calculation.
            const DTYPE_GPU fwd_val = ts_alpha_in[state];

            // Calculate fwd/bwd guide product in log space.
            const DTYPE_GPU val = fwd_val + bwd[ts_idx + state];

            fwd_vals[state] = val;
            atomicMaxFloat(&max_val, val);
        }
        exp_sum = 0.0;
        __syncthreads();

        // enter exp vals
        for (uint64_t state = state_begin; state < state_end; ++state) {
            DTYPE_GPU exp_val = __expf(fwd_vals[state] - max_val);
            exp_vals[state] = exp_val;
            atomicAdd(&exp_sum, exp_val);
        }
        __syncthreads();

        // calculate posterior probability
        for (uint64_t state = state_begin; state < state_end; ++state) {
            out[ts_idx + state] = exp_vals[state] / exp_sum;
        }
        max_val = FLT_MIN;
        __syncthreads();
    }
}

void decode_gpu(
    const int T,
    const int N,
    const int C,
    const int target_threads,
    float *scores_TNC,
    std::vector<DecodedChunk>& chunk_results,
    const int state_len,
    const DecoderOptions* options
) {
    const int n_base = 4;
    const int num_states = std::pow(n_base, state_len);

    // calculate grid / block dims
    const int target_block_width = (int)ceil(sqrt((float)num_states));
    int block_width = 2;
    int grid_len = 2;
    while (block_width < target_block_width) {
        block_width *= 2;
    }
    while (grid_len < N) {
        grid_len *= 2;
    }

    fprintf(stderr, "chosen block_dims: %d x %d for num_states %d\n", block_width, block_width, num_states);
    fprintf(stderr, "chosen grid_len: %d for batch size %d\n", grid_len, N);

    double t0, t1, elapsed;
    dim3 block_size(block_width, block_width, 1);
	dim3 grid_size(grid_len, 1, 1);

    // expect input already transformed
    // scores_TNC = scores_TNC.to(torch::kCPU).to(DTYPE_GPU).transpose(0, 1).contiguous();
    
    
    const int states_per_thread = std::max(1, num_states / (block_width * block_width));
    const uint64_t num_scan_elem = N * (T + 1) * num_states;

    LOG_TRACE("scores tensor dim: %d, %d, %d", T, N, C);

    DTYPE_GPU *bwd_NTC = (DTYPE_GPU *)malloc(num_scan_elem * sizeof(DTYPE_GPU));
    MALLOC_CHK(bwd_NTC);
    DTYPE_GPU *post_NTC = (DTYPE_GPU *)malloc(num_scan_elem * sizeof(DTYPE_GPU));
    MALLOC_CHK(post_NTC);

    DTYPE_GPU *scores_TNC_cuda;
    DTYPE_GPU *bwd_NTC_cuda;
    DTYPE_GPU *post_NTC_cuda;

    // copy score tensor over
    hipMalloc((void **)&scores_TNC_cuda, sizeof(DTYPE_GPU) * T * N * C);
	checkCudaError();
	hipMemcpy(scores_TNC_cuda, scores_TNC, sizeof(DTYPE_GPU) * T * N * C, hipMemcpyHostToDevice);
	checkCudaError();

    // init scan tensors
    hipMalloc((void **)&bwd_NTC_cuda, sizeof(DTYPE_GPU) * num_scan_elem);
	checkCudaError();
    hipMalloc((void **)&post_NTC_cuda, sizeof(DTYPE_GPU) * num_scan_elem);
	checkCudaError();

#ifdef BENCH
    int n_batch = 140; // simulate 20k reads
    if (num_states == 64) n_batch = 140; // fast
    else if (num_states == 256) n_batch = 345; // hac
    else if (num_states == 1024) n_batch = 685; // sup
    fprintf(stderr, "simulating %d batches...\n", n_batch);
#endif

    // bwd scan
	t0 = realtime();
#ifdef BENCH
    for (int i = 0; i < n_batch; ++i)
#endif
    {
        bwd_scan<<<grid_size,block_size>>>(scores_TNC_cuda, bwd_NTC_cuda, T, N, num_states, states_per_thread);
        hipDeviceSynchronize();
        checkCudaError();
    }
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    fprintf(stderr, "bwd scan completed in %f secs\n", elapsed);
    
    // fwd + post scan
	t0 = realtime();
#ifdef BENCH
    for (int i = 0; i < n_batch; ++i)
#endif
    {
        fwd_post_scan<<<grid_size,block_size>>>(scores_TNC_cuda, bwd_NTC_cuda, post_NTC_cuda, T, N, num_states, states_per_thread);
        hipDeviceSynchronize();
        checkCudaError();
    }
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    fprintf(stderr, "fwd scan completed in %f secs\n", elapsed);

	// copy results
    hipMemcpy(bwd_NTC, bwd_NTC_cuda, sizeof(DTYPE_GPU) * num_scan_elem, hipMemcpyDeviceToHost);
    checkCudaError();
	hipMemcpy(post_NTC, post_NTC_cuda, sizeof(DTYPE_GPU) * num_scan_elem, hipMemcpyDeviceToHost);
    checkCudaError();

    // write tensors
    FILE *fp;
    fp = fopen("scores_TNC.blob", "w");
    fwrite(scores_TNC, sizeof(DTYPE_GPU), T * N * C, fp);
    fclose(fp);

    fp = fopen("bwd_NTC.blob", "w");
    fwrite(bwd_NTC, sizeof(DTYPE_GPU), num_scan_elem, fp);
    fclose(fp);

    fp = fopen("post_NTC.blob", "w");
    fwrite(post_NTC, sizeof(DTYPE_GPU), num_scan_elem, fp);
    fclose(fp);

    // cleanup
    free(bwd_NTC);
    free(post_NTC);
    
    hipFree(scores_TNC_cuda);
    hipFree(bwd_NTC_cuda);
    hipFree(post_NTC_cuda);
}
