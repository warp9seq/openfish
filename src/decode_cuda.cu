#include "hip/hip_runtime.h"
#include "decode_cuda.h"
#include "scan_cuda.cuh"
#include "beam_search_cuda.cuh"
#include "error.h"
#include "cuda_utils.cuh"

#include <openfish/openfish_error.h>

#include <flash.h>

void run_flash(
    void *q,
    void *k,
    void *v,
    void **o
) {
    int batch_stride = 426496;
    int head_stride = 53312;
    int row_stride = 64;
    size_t batch_size = 512;
    size_t seqlen = 833;
    size_t num_heads = 8;
    size_t head_dim = 64;
    size_t numel = batch_size * seqlen * num_heads * head_dim;

    cutlass::half_t *q_gpu;
    cutlass::half_t *k_gpu;
    cutlass::half_t *v_gpu;
    cutlass::half_t *o_gpu;
    hipMalloc((void **)&q_gpu, sizeof(cutlass::half_t) * numel);
	checkCudaError();
    hipMalloc((void **)&k_gpu, sizeof(cutlass::half_t) * numel);
	checkCudaError();
    hipMalloc((void **)&v_gpu, sizeof(cutlass::half_t) * numel);
	checkCudaError();

    hipMemcpy(q_gpu, q, sizeof(cutlass::half_t) * numel, hipMemcpyHostToDevice);
    checkCudaError();
    hipMemcpy(k_gpu, k, sizeof(cutlass::half_t) * numel, hipMemcpyHostToDevice);
    checkCudaError();
    hipMemcpy(v_gpu, v, sizeof(cutlass::half_t) * numel, hipMemcpyHostToDevice);
    checkCudaError();
    
    *o = (uint8_t *)malloc(sizeof(cutlass::half_t) * numel);
    MALLOC_CHK(*o);

    hipMalloc((void **)&o_gpu, sizeof(cutlass::half_t) * numel);
	checkCudaError();

    int seqlen_q = seqlen;
    int seqlen_k = seqlen;
    int num_heads_k = num_heads;
    
    int q_batch_stride = batch_stride;
    int k_batch_stride = batch_stride;
    int v_batch_stride = batch_stride;
    int o_batch_stride = batch_stride;
    int q_head_stride = head_stride;
    int k_head_stride = head_stride;
    int v_head_stride = head_stride;
    int o_head_stride = head_stride;
    int q_row_stride = row_stride;
    int k_row_stride = row_stride;
    int v_row_stride = row_stride;
    int o_row_stride = row_stride;
    float softmax_scale = 1.0 / std::sqrt(num_heads);
    int window_size_left = 127;
    int window_size_right = 128;
    bool casual = false;

    // upload qkv
    flash_attn::flash_attention_forward(
        q_gpu,
        k_gpu,
        v_gpu,
        o_gpu,
        batch_size,
        seqlen_q,
        seqlen_k,
        num_heads,
        num_heads_k,
        head_dim,
        q_batch_stride,
        k_batch_stride,
        v_batch_stride,
        o_batch_stride,
        q_head_stride,
        k_head_stride,
        v_head_stride,
        o_head_stride,
        q_row_stride,
        k_row_stride,
        v_row_stride,
        o_row_stride,
        softmax_scale,
        casual,
        window_size_left,
        window_size_right
    );

    hipMemcpy(*o, o_gpu, sizeof(cutlass::half_t) * numel, hipMemcpyDeviceToHost);
    checkCudaError();
}

openfish_gpubuf_t *gpubuf_init_cuda(
    const int T,
    const int N,
    const int state_len
) {
    openfish_gpubuf_t *gpubuf = (openfish_gpubuf_t *)(malloc(sizeof(openfish_gpubuf_t)));
    MALLOC_CHK(gpubuf);

    const int num_states = pow(NUM_BASES, state_len);

    // scan tensors
    hipMalloc((void **)&gpubuf->bwd_NTC, sizeof(float) * N * (T + 1) * num_states);
	checkCudaError();
    hipMalloc((void **)&gpubuf->post_NTC, sizeof(float) * N * (T + 1) * num_states);
	checkCudaError();

    // return buffers
    hipMalloc((void **)&gpubuf->moves, sizeof(uint8_t) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->sequence, sizeof(char) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->qstring, sizeof(char) * N * T);
    checkCudaError();

    // beamsearch buffers
    hipMalloc((void **)&gpubuf->beam_vector, sizeof(beam_element_t) * N * MAX_BEAM_WIDTH * (T + 1));
    checkCudaError();
    hipMalloc((void **)&gpubuf->states, sizeof(state_t) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->qual_data, sizeof(float) * N * T * NUM_BASES);
    checkCudaError();
    hipMalloc((void **)&gpubuf->base_probs, sizeof(float) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->total_probs, sizeof(float) * N * T);
    checkCudaError();

    return gpubuf;
}

void gpubuf_free_cuda(
    openfish_gpubuf_t *gpubuf
) {
    hipFree(gpubuf->bwd_NTC);
    checkCudaError();
    hipFree(gpubuf->post_NTC);
    checkCudaError();

    hipFree(gpubuf->moves);
    checkCudaError();
    hipFree(gpubuf->sequence);
    checkCudaError();
    hipFree(gpubuf->qstring);
    checkCudaError();

    hipFree(gpubuf->beam_vector);
    checkCudaError();
    hipFree(gpubuf->states);
    checkCudaError();
    hipFree(gpubuf->qual_data);
    checkCudaError();
    hipFree(gpubuf->base_probs);
    checkCudaError();
    hipFree(gpubuf->total_probs);
    checkCudaError();

    free(gpubuf);
}

void decode_cuda(
    const int T,
    const int N,
    const int C,
    void *scores_TNC,
    const int state_len,
    const openfish_opt_t *options,
    const openfish_gpubuf_t *gpubuf,
    uint8_t **moves,
    char **sequence,
    char **qstring
) {
    const int num_states = pow(NUM_BASES, state_len);

    // calculate grid / block dims
    const int target_block_width = (int)ceil(sqrt((float)num_states));
    int block_width = 2;
    while (block_width < target_block_width) {
        block_width *= 2;
    }

    OPENFISH_LOG_TRACE("chosen block_dims: %d x %d for num_states %d", block_width, block_width, num_states);
    
    dim3 block_size(block_width, block_width, 1);
    dim3 block_size_beam(MAX_BEAM_WIDTH * NUM_BASES, 1, 1);
    dim3 block_size_gen(1, 1, 1);
	dim3 grid_size(N, 1, 1);

    OPENFISH_LOG_TRACE("scores tensor dim: %d, %d, %d", T, N, C);

    scan_args_t scan_args = {0};
    scan_args.scores_in = scores_TNC;
    scan_args.T = T;
    scan_args.N = N;
    scan_args.C = C;
    scan_args.num_states = num_states;
    scan_args.fixed_stay_score = options->blank_score;

    // init results
    *moves = (uint8_t *)malloc(N * T * sizeof(uint8_t));
    MALLOC_CHK(*moves);
    *sequence = (char *)malloc(N * T * sizeof(char));
    MALLOC_CHK(*sequence);
    *qstring = (char *)malloc(N * T * sizeof(char));
    MALLOC_CHK(*qstring);

    hipMemset(gpubuf->moves, 0, sizeof(uint8_t) * N * T);
	checkCudaError();
    hipMemset(gpubuf->sequence, 0, sizeof(char) * N * T);
	checkCudaError();
    hipMemset(gpubuf->qstring, 0, sizeof(char) * N * T);
	checkCudaError();

    const int num_state_bits = (int)log2((double)num_states);
    const float fixed_stay_score = options->blank_score;
    const float q_scale = options->q_scale;
    const float q_shift = options->q_shift;
    const float beam_cut = options->beam_cut;

    beam_args_t beam_args = {0};
    beam_args.scores_TNC = (half *)scores_TNC;
    beam_args.bwd_NTC = gpubuf->bwd_NTC;
    beam_args.post_NTC = gpubuf->post_NTC;
    beam_args.T = T;
    beam_args.N = N;
    beam_args.C = C;
    beam_args.num_state_bits = num_state_bits;

    // bwd scan
    // fwd + post scan
    // beam search
    bwd_scan<<<grid_size,block_size>>>(scan_args, gpubuf->bwd_NTC);
    checkCudaError();
    hipDeviceSynchronize();
    checkCudaError();

    beam_search<<<grid_size,block_size_beam>>>(
        beam_args,
        (state_t *)gpubuf->states,
        gpubuf->moves,
        (beam_element_t *)gpubuf->beam_vector,
        beam_cut,
        fixed_stay_score,
        1.0f
    );
    checkCudaError();
    hipDeviceSynchronize();
    checkCudaError();

    fwd_post_scan<<<grid_size,block_size>>>(scan_args, gpubuf->bwd_NTC, gpubuf->post_NTC);
    checkCudaError();
    hipDeviceSynchronize();
    checkCudaError();

    compute_qual_data<<<grid_size,block_size_gen>>>(
        beam_args,
        (state_t *)gpubuf->states,
        gpubuf->qual_data,
        1.0f
    );
    checkCudaError();
    hipDeviceSynchronize();
    checkCudaError();
    
    generate_sequence<<<grid_size,block_size_gen>>>(
        beam_args,
        gpubuf->moves,
        (state_t *)gpubuf->states,
        gpubuf->qual_data,
        gpubuf->base_probs,
        gpubuf->total_probs,
        gpubuf->sequence,
        gpubuf->qstring,
        q_shift,
        q_scale
    );
    checkCudaError();
    hipDeviceSynchronize();
    checkCudaError();

    // copy beam_search results
    hipMemcpy(*moves, gpubuf->moves, sizeof(uint8_t) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();
	hipMemcpy(*sequence, gpubuf->sequence, sizeof(char) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();
    hipMemcpy(*qstring, gpubuf->qstring, sizeof(char) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();
}

// misc stuff for testing //////////////////////////////////////////////////////
void set_device_cuda(
    int device
) {
    hipSetDevice(device);
	checkCudaError();
}

void *upload_scores_to_cuda(
    const int T,
    const int N,
    const int C,
    const void *scores_TNC
) {
    void *scores_TNC_gpu;

    hipMalloc((void **)&scores_TNC_gpu, sizeof(half) * T * N * C);
	checkCudaError();

	hipMemcpy(scores_TNC_gpu, scores_TNC, sizeof(half) * T * N * C, hipMemcpyHostToDevice);
	checkCudaError();

    return scores_TNC_gpu;
}

void free_scores_cuda(
    void *scores_TNC_gpu
) {
    hipFree(scores_TNC_gpu);
	checkCudaError();
}

void write_gpubuf_cuda(
    const uint64_t T,
    const uint64_t N,
    const int state_len,
    const openfish_gpubuf_t *gpubuf
) {
    const int num_states = pow(NUM_BASES, state_len);

    float *bwd_NTC = (float *)malloc(N * (T + 1) * num_states * sizeof(float));
    MALLOC_CHK(bwd_NTC);
    float *post_NTC = (float *)malloc(N * (T + 1) * num_states * sizeof(float));
    MALLOC_CHK(post_NTC);
    state_t *states = (state_t *)malloc(N * T * sizeof(state_t));
    MALLOC_CHK(states);
    float *qual_data = (float *)malloc(N * T * NUM_BASES * sizeof(float));
    MALLOC_CHK(qual_data);
    float *base_probs = (float *)malloc(N * T * sizeof(float));
    MALLOC_CHK(base_probs);
    float *total_probs = (float *)malloc(N * T * sizeof(float));
    MALLOC_CHK(total_probs);

    // copy scan results
    hipMemcpy(bwd_NTC, gpubuf->bwd_NTC, sizeof(float) * N * (T + 1) * num_states, hipMemcpyDeviceToHost);
    checkCudaError();
	hipMemcpy(post_NTC, gpubuf->post_NTC, sizeof(float) * N * (T + 1) * num_states, hipMemcpyDeviceToHost);
    checkCudaError();

    // copy intermediate
    hipMemcpy(states, gpubuf->states, sizeof(state_t) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();

    hipMemcpy(total_probs, gpubuf->total_probs, sizeof(float) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();

    hipMemcpy(qual_data, gpubuf->qual_data, sizeof(float) * N * T * NUM_BASES, hipMemcpyDeviceToHost);
    checkCudaError();

    hipMemcpy(base_probs, gpubuf->base_probs, sizeof(float) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();

    // write results
    FILE *fp;

    fp = fopen("bwd_NTC.blob", "w");
    F_CHK(fp, "bwd_NTC.blob");
    if (fwrite(bwd_NTC, sizeof(float), N * (T + 1) * num_states, fp) != N * (T + 1) * num_states) {
        fprintf(stderr, "error writing sequence file: %s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }
    fclose(fp);

    fp = fopen("post_NTC.blob", "w");
    F_CHK(fp, "post_NTC.blob");
    if (fwrite(post_NTC, sizeof(float), N * (T + 1) * num_states, fp) != N * (T + 1) * num_states) {
        fprintf(stderr, "error writing sequence file: %s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }
    fclose(fp);

    // write beam results
    fp = fopen("qual_data.blob", "w");
    F_CHK(fp, "qual_data.blob");
    if (fwrite(qual_data, sizeof(float), N * T * NUM_BASES, fp) != N * T * NUM_BASES) {
        fprintf(stderr, "error writing sequence file: %s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }
    fclose(fp);

    fp = fopen("base_probs.blob", "w");
    F_CHK(fp, "base_probs.blob");
    if (fwrite(base_probs, sizeof(float), N * T, fp) != N * T) {
        fprintf(stderr, "error writing sequence file: %s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }
    fclose(fp);

    fp = fopen("total_probs.blob", "w");
    F_CHK(fp, "total_probs.blob");
    if (fwrite(total_probs, sizeof(float), N * T, fp) != N * T) {
        fprintf(stderr, "error writing sequence file: %s\n", strerror(errno));
        exit(EXIT_FAILURE);
    }
    fclose(fp);

    // cleanup
    free(bwd_NTC);
    free(post_NTC);
    free(states);
    free(qual_data);
    free(base_probs);
    free(total_probs);
}
////////////////////////////////////////////////////////////////////////////////