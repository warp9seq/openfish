#include "hip/hip_runtime.h"
#include "decode_cuda.h"
#include "scan_cuda.cuh"
#include "beam_search_cuda.cuh"
#include "error.h"
#include "cuda_utils.cuh"
#include "misc.h"

#include <openfish/openfish_error.h>

#include <hip/hip_fp16.h>

void *upload_scores_to_cuda(
    const int T,
    const int N,
    const int C,
    const void *scores_TNC
) {
    void *scores_TNC_gpu;

    hipMalloc((void **)&scores_TNC_gpu, sizeof(half) * T * N * C);
	checkCudaError();

	hipMemcpy(scores_TNC_gpu, scores_TNC, sizeof(half) * T * N * C, hipMemcpyHostToDevice);
	checkCudaError();

    return scores_TNC_gpu;
}

void free_scores_cuda(
    void *scores_TNC_gpu
) {
    hipFree(scores_TNC_gpu);
	checkCudaError();
}

openfish_gpubuf_t *gpubuf_init_cuda(
    const int T,
    const int N,
    const int state_len
) {
    openfish_gpubuf_t *gpubuf = (openfish_gpubuf_t *)(malloc(sizeof(openfish_gpubuf_t)));

    const int num_states = pow(NUM_BASES, state_len);

    // scan tensors
    hipMalloc((void **)&gpubuf->bwd_NTC, sizeof(float) *  N * (T + 1) * num_states);
	checkCudaError();
    hipMalloc((void **)&gpubuf->post_NTC, sizeof(float) *  N * (T + 1) * num_states);
	checkCudaError();

    // return buffers
    hipMalloc((void **)&gpubuf->moves, sizeof(uint8_t) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->sequence, sizeof(char) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->qstring, sizeof(char) * N * T);
    checkCudaError();

    // beamsearch buffers
    hipMalloc((void **)&gpubuf->beam_vector, sizeof(beam_element_t) * N * MAX_BEAM_WIDTH * (T + 1));
    checkCudaError();
    hipMalloc((void **)&gpubuf->states, sizeof(state_t) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->qual_data, sizeof(float) * N * T * NUM_BASES);
    checkCudaError();
    hipMalloc((void **)&gpubuf->base_probs, sizeof(float) * N * T);
    checkCudaError();
    hipMalloc((void **)&gpubuf->total_probs, sizeof(float) * N * T);
    checkCudaError();

    return gpubuf;
}

void gpubuf_free_cuda(
    openfish_gpubuf_t *gpubuf
) {
    hipFree(gpubuf->bwd_NTC);
    checkCudaError();
    hipFree(gpubuf->post_NTC);
    checkCudaError();

    hipFree(gpubuf->moves);
    checkCudaError();
    hipFree(gpubuf->sequence);
    checkCudaError();
    hipFree(gpubuf->qstring);
    checkCudaError();

    hipFree(gpubuf->beam_vector);
    checkCudaError();
    hipFree(gpubuf->states);
    checkCudaError();
    hipFree(gpubuf->qual_data);
    checkCudaError();
    hipFree(gpubuf->base_probs);
    checkCudaError();
    hipFree(gpubuf->total_probs);
    checkCudaError();

    free(gpubuf);
}

void decode_cuda(
    const int T,
    const int N,
    const int C,
    void *scores_TNC,
    const int state_len,
    const openfish_opt_t *options,
    const openfish_gpubuf_t *gpubuf,
    uint8_t **moves,
    char **sequence,
    char **qstring
) {
    const int num_states = pow(NUM_BASES, state_len);

    // calculate grid / block dims
    const int target_block_width = (int)ceil(sqrt((float)num_states));
    int block_width = 2;
    int grid_len = 2;
    while (block_width < target_block_width) {
        block_width *= 2;
    }
    while (grid_len < N) {
        grid_len *= 2;
    }

    OPENFISH_LOG_DEBUG("chosen block_dims: %d x %d for num_states %d", block_width, block_width, num_states);
    OPENFISH_LOG_DEBUG("chosen grid_len: %d for batch size %d", grid_len, N);

    double t0, t1, elapsed;
    dim3 block_size(block_width, block_width, 1);
    dim3 block_size_beam(MAX_BEAM_WIDTH, 1, 1);
    dim3 block_size_gen(1, 1, 1);
	dim3 grid_size(grid_len, 1, 1);

    OPENFISH_LOG_DEBUG("scores tensor dim: %d, %d, %d", T, N, C);

    scan_args_t scan_args = {0};
    scan_args.scores_in = scores_TNC;
    scan_args.T = T;
    scan_args.N = N;
    scan_args.C = C;
    scan_args.num_states = num_states;
    scan_args.fixed_stay_score = options->blank_score;

    // bwd scan
	t0 = realtime();
    bwd_scan<<<grid_size,block_size>>>(scan_args, gpubuf->bwd_NTC);
    hipDeviceSynchronize();
    checkCudaError();
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    OPENFISH_LOG_DEBUG("bwd scan completed in %f secs", elapsed);
    
    // fwd + post scan
	t0 = realtime();
    fwd_post_scan<<<grid_size,block_size>>>(scan_args, gpubuf->bwd_NTC, gpubuf->post_NTC);
    hipDeviceSynchronize();
    checkCudaError();
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    OPENFISH_LOG_DEBUG("fwd scan completed in %f secs", elapsed);

    // beam search

    // init results
    *moves = (uint8_t *)malloc(N * T * sizeof(uint8_t));
    MALLOC_CHK(*moves);
    *sequence = (char *)malloc(N * T * sizeof(char));
    MALLOC_CHK(*sequence);
    *qstring = (char *)malloc(N * T * sizeof(char));
    MALLOC_CHK(*qstring);

    hipMemset(gpubuf->moves, 0, sizeof(uint8_t) * N * T);
	checkCudaError();
    hipMemset(gpubuf->sequence, 0, sizeof(char) * N * T);
	checkCudaError();
    hipMemset(gpubuf->qstring, 0, sizeof(char) * N * T);
	checkCudaError();

    const int num_state_bits = (int)log2(num_states);
    const float fixed_stay_score = options->blank_score;
    const float q_scale = options->q_scale;
    const float q_shift = options->q_shift;
    const float beam_cut = options->beam_cut;

    beam_args_t beam_args = {0};
    beam_args.scores_TNC = (half *)scores_TNC;
    beam_args.bwd_NTC = gpubuf->bwd_NTC;
    beam_args.post_NTC = gpubuf->post_NTC;
    beam_args.T = T;
    beam_args.N = N;
    beam_args.C = C;
    beam_args.num_state_bits = num_state_bits;

    t0 = realtime();
    beam_search<<<grid_size,block_size_beam>>>(
        beam_args,
        (state_t *)gpubuf->states,
        gpubuf->moves,
        (beam_element_t *)gpubuf->beam_vector,
        beam_cut,
        fixed_stay_score,
        1.0f
    );
    hipDeviceSynchronize();
    checkCudaError();
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    OPENFISH_LOG_DEBUG("beam search completed in %f secs", elapsed);

    t0 = realtime();
    compute_qual_data<<<grid_size,block_size_gen>>>(
        beam_args,
        (state_t *)gpubuf->states,
        gpubuf->qual_data,
        1.0f
    );
    hipDeviceSynchronize();
    checkCudaError();
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    OPENFISH_LOG_DEBUG("compute quality data completed in %f secs", elapsed);

    t0 = realtime();
    generate_sequence<<<grid_size,block_size_gen>>>(
        beam_args,
        gpubuf->moves,
        (state_t *)gpubuf->states,
        gpubuf->qual_data,
        gpubuf->base_probs,
        gpubuf->total_probs,
        gpubuf->sequence,
        gpubuf->qstring,
        q_shift,
        q_scale
    );
    hipDeviceSynchronize();
    checkCudaError();
	// end timing
	t1 = realtime();
    elapsed = t1 - t0;
    OPENFISH_LOG_DEBUG("generate sequence completed in %f secs", elapsed);

    // copy beam_search results
    hipMemcpy(*moves, gpubuf->moves, sizeof(uint8_t) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();
	hipMemcpy(*sequence, gpubuf->sequence, sizeof(char) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();
    hipMemcpy(*qstring, gpubuf->qstring, sizeof(char) * N * T, hipMemcpyDeviceToHost);
    checkCudaError();
}
